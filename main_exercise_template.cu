#include "hip/hip_runtime.h"
#include "Image.h"
#include "PPM.h"

#include <cstdio>
#include <cassert>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

static void CheckCudaErrorAux(const char *, unsigned, const char *,
		hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
		const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "("
			<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}

// useful defines
#define Mask_width 5
#define Mask_radius Mask_width / 2
#define TILE_WIDTH 16
#define w (TILE_WIDTH + Mask_width - 1)
#define clamp(x) (min(max((x), 0.0), 1.0))

// TODO INSERT CODE HERE
__global__ void convolution(float *I, const float *__restrict__ M, float *P,
		int channels, int width, int height) {
			int maskWidth = 5;
		 	int maskRadius = maskWidth / 2;
		 	int xOffset, yOffset;
		 	float accum = 0.0;
		 	int i = blockIdx.y * blockDim.y + threadIdx.y;
		 	int j = blockIdx.x * blockDim.x + threadIdx.x;
			int x,y;
			for (int k = 0; k < channels; k++) {
				for (y = 0; y < Mask_width; y++){
					for (x = 0; x < Mask_width; x++){
						yOffset = i+y;
						xOffset = j+x;
						if (xOffset >= 0
								&& xOffset < width
								&& yOffset >= 0
								&& yOffset < height) {
	          	float imagePixel = I[(yOffset * width + xOffset) * channels + k];
	          	float maskValue = M[(y+maskRadius)*maskWidth+x+maskRadius];
	          	accum += imagePixel * maskValue;
	          }
					}

				if (y < height && x < width)
					P[(y * width + x) * channels + k] = clamp(accum);
				}
			}

}

// simple test to read/write PPM images, and process Image_t data
void test_images() {
	Image_t* inputImg = PPM_import("computer_programming.ppm");
	for (int i = 0; i < 300; i++) {
		Image_setPixel(inputImg, i, 100, 0, float(i) / 300);
		Image_setPixel(inputImg, i, 100, 1, float(i) / 300);
		Image_setPixel(inputImg, i, 100, 2, float(i) / 200);
	}
	PPM_export("test_output.ppm", inputImg);
	Image_t* newImg = PPM_import("test_output.ppm");
	inputImg = PPM_import("computer_programming.ppm");
	if (Image_is_same(inputImg, newImg))
		printf("Img uguali\n");
	else
		printf("Img diverse\n");
}

int main() {
	const int maskRows = 5;
	const int maskColumns = 5;
	int imageChannels;
	int imageWidth;
	int imageHeight;
	Image_t* inputImage;
	Image_t* outputImage;
	float *hostInputImageData;
	float *hostOutputImageData;
	float *deviceInputImageData;
	float *deviceOutputImageData;
	float *deviceMaskData;
	float hostMaskData[maskRows * maskColumns] = { 0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04,
			0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04, 0.04,
			0.04, 0.04, 0.04, 0.04, 0.04, 0.04, };
	test_images();
	inputImage = PPM_import("computer_programming.ppm");

	assert(maskRows == 5); /* mask height is fixed to 5 in this exercise */
	assert(maskColumns == 5); /* mask width is fixed to 5 in this exercise */

	imageWidth = Image_getWidth(inputImage);
	printf("Image Width %i \n", imageWidth);
	imageHeight = Image_getHeight(inputImage);
	printf("Image Height %i \n", imageHeight);
	imageChannels = Image_getChannels(inputImage);
	printf("Image Channels %i \n", imageChannels);

	outputImage = Image_new(imageWidth, imageHeight, imageChannels);

	hostInputImageData = Image_getData(inputImage);
	hostOutputImageData = Image_getData(outputImage);

	// Allocate device buffers
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	cudaStatus = hipMalloc((void**)&deviceInputImageData,imageWidth*imageHeight*imageChannels* sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&deviceOutputImageData,imageWidth*imageHeight*imageChannels* sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&deviceMaskData,maskRows*maskColumns* sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	// Copy memory from host to device
	CUDA_CHECK_RETURN(hipMemcpy(deviceInputImageData, hostInputImageData,sizeof(float) *imageWidth * imageHeight * imageChannels,hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(deviceMaskData, hostMaskData,maskRows*maskColumns* sizeof(float),hipMemcpyHostToDevice));
	//Grid and Block
	dim3 dimGrid(ceil((float)imageWidth/TILE_WIDTH), ceil((float)imageHeight/TILE_WIDTH));
	dim3 dimBlock(TILE_WIDTH,TILE_WIDTH,1);
	convolution<<<dimGrid, dimBlock>>>(deviceInputImageData, deviceMaskData,
			deviceOutputImageData, imageChannels, imageWidth, imageHeight);

	// Copy from device to host memory
	CUDA_CHECK_RETURN(hipMemcpy(hostOutputImageData,
               deviceOutputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyDeviceToHost));

	PPM_export("processed_computer_programming.ppm", outputImage);

	// Free device memory
	hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);
  hipFree(deviceMaskData);

	Image_delete(outputImage);
	Image_delete(inputImage);

	return 0;
}
